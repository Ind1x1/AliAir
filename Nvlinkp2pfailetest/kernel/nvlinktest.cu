#include "hip/hip_runtime.h"
/*Copy right
*   Leyi Ye 2025 (alpha) poject
*   This project is used to test the nvlink p2p bandwidth and latenc
*/
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>


#undef CUDA_CHECK
#define CUDA_CHECK(stmt)                                                          \
    do {                                                                          \
        hipError_t result = (stmt);                                              \
        if (hipSuccess != result) {                                              \
            fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                    hipGetErrorString(result));                                  \
            exit(-1);                                                             \
        }                                                                         \
    } while (0)


#define cudaCheckError()                                                                     \
    {                                                                                        \
        hipError_t e = hipGetLastError();                                                  \
        if (e != hipSuccess) {                                                              \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
            exit(EXIT_FAILURE);                                                              \
        }                                                                                    \
    }



__global__ void Ring_Shift_test(int *destination) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    nvshmem_int_p(destination, mype, peer);
}

void performRing_Shift_test(int *destination, int repeat)
{
    int blockSize = 0;
    int numBlocks = 0;

    hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, Ring_Shift_test);

    for(int r = 0; r < repeat; r++){
        Ring_Shift_test<<<numBlocks, blockSize>>>(destination);
    }
}

void outputNVSHMEMBandwidthRingMatrix(int numElems, int repeat)
{
    int mype_node;
    hipEvent_t start, stop;
    hipStream_t stream;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    nvshmem_init();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);
    hipStreamCreate(&stream);
    int npes = nvshmem_n_pes();

    int *destination = (int *) nvshmem_malloc(sizeof(int) * numElems);
    cudaCheckError();

    hipEventRecord(start, stream);
    cudaCheckError();
    
    performRing_Shift_test(destination, repeat);
    nvshmemx_barrier_all_on_stream(stream);

    hipEventRecord(stop, stream);
    cudaCheckError();

    float time_ms;
    hipEventElapsedTime(&time_ms, start, stop);
    double time_s = time_ms / 1e3;

    double gb = numElems * sizeof(int) * repeat / (double)1e9;
    double bandwidth = gb / time_s;


    printf("GPU NVSHMEM Bandwidth Ring Matrix\n");
    printf("----------------------------------\n");
    printf("%6.02f", bandwidth);
    

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);

    nvshmem_free(destination);
    nvshmem_finalize();
    cudaCheckError();
}

int main()
{
    int numGPUs,numElems = 4000000;
    hipGetDeviceCount(&numGPUs);
    cudaCheckError();

    outputNVSHMEMBandwidthRingMatrix(numElems, 5);
}